#include "hip/hip_runtime.h"
%%writefile WarpSynchronization.cu

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <math.h>

#define BLOCK_SIZE 64

__global__ void syncWarp_test()
{
    int tID = threadIdx.x;
    int warpID = (int) (tID / 32);
    __shared__ int masterID[BLOCK_SIZE/32];

    if (threadIdx.x % 32 == 0) {
        masterID[warpID] = tID;
    }
    __syncwarp(); // intra-warp synchronization (barrier)

    printf("[T%d] The master of our warp is %d\n", tID, masterID [warpID]);
}

int main() {
    syncWarp_test <<< 1, BLOCK_SIZE >>> ();
    hipDeviceSynchronize();  // Ensure the kernel completes before the program exits
    
    hipError_t err = hipGetLastError();  // Check for kernel launch errors
    if (err != hipSuccess) {
        printf("CUDA Error: %s\n", hipGetErrorString(err));
    }
    
    return 0;
}